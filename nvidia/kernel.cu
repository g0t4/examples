
#include <hip/hip_runtime.h>
extern "C" __global__ void add(int *out, const int *a, const int *b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = a[i] + b[i];
}
